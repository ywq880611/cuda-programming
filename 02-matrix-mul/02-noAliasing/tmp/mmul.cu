
#include <hip/hip_runtime.h>
#include <stdio.h>

const int N = 1 << 10;
const int bytes = N * N * sizeof(N);
int h_a[N * N];
int h_b[N * N];
int h_c[N * N];

// On RTX 3090, the if we use tmp to avoid alias (or avoid to access cache), it
// shows `25ms vs 10ms`. If we also add `__restrict__` here, we couldn't see
// more benefits, so I guess `tmp` do same thing as `__restrict__`, maybe I
// could check PTX later?
__global__ void matrixMul(int* a, int* b, int* c, int N) {
  // take N as both no. of rows and coulums, so here is
  // a sqare matrix.

  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  int tmp = 0;
  for (int i = 0; i < N; i++) {
    tmp += a[row * N + i] * b[N * i + col];
  }
  c[row * N + col] = tmp;
}

void verify_results(int* a, int* b, int* c, int N) {
  for (int row = 0; row < N; row++) {
    for (int col = 0; col < N; col++) {
      int a_times_b = 0;
      for (int i = 0; i < N; i++) {
        a_times_b += a[row * N + i] * b[N * i + col];
      }
      if (a_times_b != c[row * N + col]) {
        printf("the result is wrong at row: %d, column: %d\n", row, col);
        printf("it should be %d, but it's %d\n", a_times_b, c[row * N + col]);
        abort();
      }
    }
  }
}

int main() {
  // Initialize h_a and h_b firstly.
  for (int row = 0; row < N; row++) {
    for (int col = 0; col < N; col++) {
      h_a[row * N + col] = 1;
      h_b[row * N + col] = 1;
      // h_a[row * N + col] = rand() % 100;
      // h_b[row * N + col] = rand() % 100;
    }
  }

  int* d_a;
  int* d_b;
  int* d_c;
  hipMalloc(&d_a, bytes);
  hipMalloc(&d_b, bytes);
  hipMalloc(&d_c, bytes);

  hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

  const int THREAD = 32;
  const int BLOCK = (N + THREAD - 1) / THREAD;

  const dim3 threads(THREAD, THREAD);
  const dim3 blocks(BLOCK, BLOCK);

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // Record start event
  hipEventRecord(start);

  for (int i = 0; i < 10; i++) {
    matrixMul<<<blocks, threads>>>(d_a, d_b, d_c, N);
  }

  // Record stop event
  hipEventRecord(stop);

  hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  printf("Kernel execution time: %f ms\n", milliseconds);

  verify_results(h_a, h_b, h_c, N);

  printf("COMPLETED SUCCESSFULLY\n");

  // Free memory on device
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  return 0;
}