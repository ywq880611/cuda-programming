#include "hip/hip_runtime.h"
#include <stdio.h>

#include <cassert>

#include "../../00-cuBLAS/mmul.cuh"

// There is just about 15% perf improve on RTX3080Ti for this case.
// I saw the GOPS improve by 2.5x, but 15% improve on kernel time.
// TODO: it maybe caused by memory bandwidth??? Check later.

#define data_type float

// Matrix dimensions, two matrixs:
// (M, K) and (K, N)
constexpr int M = 1 << 12;
constexpr int N = 1 << 12;
constexpr int K = 1 << 12;

const int THREAD_X = 32;
const int THREAD_Y = 32;

constexpr int K_stride = 32;

data_type h_a[M * K];
data_type h_b[K * N];
data_type h_c[M * N];

const int a_bytes = M * K * sizeof(data_type);
const int b_bytes = K * N * sizeof(data_type);
const int c_bytes = M * N * sizeof(data_type);

const int test_round = 100;

__global__ void matrixMul(data_type* a, data_type* b, data_type* c) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  // TODO: shared memory shape of matrix A and B shouldn't always
  // be same. We should calculate them according to blockDim.
  __shared__ data_type s_a[THREAD_Y * K_stride];
  __shared__ data_type s_b[K_stride * THREAD_X];

  data_type tmp = 0;
  for (int i = 0; i < K; i += K_stride) {
    // NOTE: the check for avoid overflow in case the K_stride is less than
    // blockDim.x pr blockDim.y
    if (threadIdx.x < K_stride) {
      s_a[threadIdx.y * K_stride + threadIdx.x] =
          a[row * K + i + threadIdx.x];
    }
    if (threadIdx.y < K_stride) {
      s_b[threadIdx.y * blockDim.x + threadIdx.x] =
          b[(i + threadIdx.y) * N + col];
    }

    __syncthreads();

    for (int j = 0; j < K_stride; j++) {
      tmp +=
          s_a[threadIdx.y * K_stride + j] * s_b[j * blockDim.x + threadIdx.x];
    }

    __syncthreads();
  }

  if (row < M && col < N) c[row * N + col] = tmp;
}

int main() {
  // Initialize h_a and h_b firstly.
  for (int row = 0; row < M; row++) {
    for (int col = 0; col < N; col++) {
      h_a[row * K + col] = rand() % 100;
      h_b[row * N + col] = rand() % 100;
    }
  }

  data_type* d_a;
  data_type* d_b;
  data_type* d_c;
  hipMalloc(&d_a, a_bytes);
  hipMalloc(&d_b, b_bytes);
  hipMalloc(&d_c, c_bytes);

  hipMemcpy(d_a, h_a, a_bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, b_bytes, hipMemcpyHostToDevice);

  // TODO: rethink the BLOCK_X and BLOCK_Y order. I thought it's
  // not important, we could switch them.
  const int BLOCK_X = N / THREAD_X;
  const int BLOCK_Y = M / THREAD_Y;

  const dim3 threads(THREAD_X, THREAD_Y);
  const dim3 blocks(BLOCK_X, BLOCK_Y);

  // NOTE: a detail, K_stride should be less or equal to blockDim.x
  // and blockDim.y, otherwise in the below loop, the s_a and s_b
  // shared memory couldn't be fully filled within blockDim.x * blockDim.y
  // threads.
  assert(K_stride <= threads.y);
  assert(K_stride <= threads.x);

  // warm up
  for (int i = 0; i < test_round; i++) {
    matrixMul<<<blocks, threads>>>(d_a, d_b, d_c);
  }

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // Record start event
  hipEventRecord(start);

  for (int i = 0; i < test_round; i++) {
    matrixMul<<<blocks, threads>>>(d_a, d_b, d_c);
  }

  // Record stop event
  hipEventRecord(stop);

  hipMemcpy(h_c, d_c, c_bytes, hipMemcpyDeviceToHost);

  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  double FLOPs = 2.0 * M * N * K * test_round;
  float GFLOPS = FLOPs / (milliseconds * 1e6);

  printf("Kernel execution time: %.02f ms\n", milliseconds);
  printf("GFLOPS: %.02f gops\n", GFLOPS);

  verify_with_cublas(M, N, K, d_a, d_b, d_c);

  printf("COMPLETED SUCCESSFULLY\n");

  // Free memory on device
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  return 0;
}