
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cassert> 

#define data_type float

// Matrix dimensions, two matrixs:
// (M, K) and (K, N)
constexpr int M = (1 << 11) + 7;
constexpr int N = (1 << 10) + 7;
constexpr int K = (1 << 11) + 7;

const int THREAD_X = 32;
const int THREAD_Y = 32;

constexpr int K_stride = 32;

// Padded matrix dimensions
constexpr int M_padded = M + (THREAD_Y - M % THREAD_Y) % THREAD_Y;
constexpr int N_padded = N + (THREAD_X - N % THREAD_X) % THREAD_X;
constexpr int K_padded = K + (K_stride - K % K_stride) % K_stride;

data_type h_a[M_padded * K_padded];
data_type h_b[K_padded * N_padded];
data_type h_c[M * N];

const int a_bytes = M_padded * K_padded * sizeof(data_type);
const int b_bytes = K_padded * N_padded * sizeof(data_type);
const int c_bytes = M * N * sizeof(data_type);

__global__ void matrixMul(data_type* a, data_type* b, data_type* c) {

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    data_type tmp = 0;
    for(int i = 0; i < K; i ++){
        tmp += a[row * K_padded + i] * b[N_padded * i + col];
    }
    if (row < M && col < N) c[row * N + col] = tmp;
}

void verify_results(data_type* a, data_type* b, data_type* c, int N){
    for(int row = 0; row < M; row ++){
        for(int col = 0; col < N; col ++){
            data_type a_times_b = 0;
            for(int i = 0; i < K; i++){
                a_times_b += a[row * K_padded + i] * b[N_padded * i + col];
            }
            if(abs(a_times_b - c[row * N + col]) > 1e-3){
                printf("the result is wrong at row: %d, column: %d\n", row, col);
                //printf("it should be %d, but it's %d\n", a_times_b, c[row * N + col]);
                abort();
            }
        }
    }
}

int main(){
    // Initialize h_a and h_b firstly.
    for(int row = 0; row < N; row ++){
        for(int col = 0; col < N; col ++){
            h_a[row * N_padded + col] = rand() % 100;
            h_b[row * N_padded + col] = rand() % 100;
        }
    }

    data_type* d_a;
    data_type* d_b;
    data_type* d_c;
    hipMalloc(&d_a, a_bytes);
    hipMalloc(&d_b, b_bytes);
    hipMalloc(&d_c, c_bytes);

    hipMemcpy(d_a, h_a, a_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, b_bytes, hipMemcpyHostToDevice);

    // TODO: rethink the BLOCK_X and BLOCK_Y order. I thought it's
    // not important, we could switch them.
    const int BLOCK_X = N_padded / THREAD_X;
    const int BLOCK_Y = M_padded / THREAD_Y;

    const dim3 threads(THREAD_X, THREAD_Y);
    const dim3 blocks(BLOCK_X, BLOCK_Y);

    // NOTE: a detail, K_stride should be less or equal to blockDim.x
    // and blockDim.y, otherwise in the below loop, the s_a and s_b
    // shared memory couldn't be fully filled within blockDim.x * blockDim.y
    // threads.
    assert(K_stride <= threads.y);
    assert(K_stride <= threads.x);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record start event
    hipEventRecord(start);

    for(int i = 0; i < 100; i ++){
        matrixMul<<<blocks, threads>>>(d_a, d_b, d_c);
    }

    // Record stop event
    hipEventRecord(stop);

    hipMemcpy(h_c, d_c, c_bytes, hipMemcpyDeviceToHost);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    long long int FLOPs = 2LL * M * N * K;
    float GFLOPS = FLOPs / (milliseconds * 1e6);

    printf("Kernel execution time: %f ms\n", milliseconds);
    printf("GFLOPS: %f gops\n", GFLOPS);

    verify_results(h_a, h_b, h_c, N);

    printf("COMPLETED SUCCESSFULLY\n");

    // Free memory on device
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}