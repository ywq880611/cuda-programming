#include "hip/hip_runtime.h"
#include <stdio.h>

#include "../../00-cuBLAS/mmul.cuh"

const int N = 1 << 12;
const int bytes = N * N * sizeof(float);
float h_a[N * N];
float h_b[N * N];
float h_c[N * N];

const int test_round = 10;

__global__ void matrixMul(float* a, float* b, float* c) {
  // take N as both no. of rows and coulums, so here is
  // a sqare matrix.

  int row = blockIdx.x * blockDim.x + threadIdx.x;
  int col = blockIdx.y * blockDim.y + threadIdx.y;

  c[row * N + col] = 0;
  for (int i = 0; i < N; i++) {
    c[row * N + col] += a[row * N + i] * b[N * i + col];
  }
}

int main() {
  // Initialize h_a and h_b firstly.
  for (int row = 0; row < N; row++) {
    for (int col = 0; col < N; col++) {
      h_a[row * N + col] = rand() % 100;
      h_b[row * N + col] = rand() % 100;
    }
  }

  float* d_a;
  float* d_b;
  float* d_c;
  hipMalloc(&d_a, bytes);
  hipMalloc(&d_b, bytes);
  hipMalloc(&d_c, bytes);

  hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

  const int THREAD = 32;
  const int BLOCK = (N + THREAD - 1) / THREAD;

  const dim3 threads(THREAD, THREAD);
  const dim3 blocks(BLOCK, BLOCK);

  // warm up
  for (int i = 0; i < test_round; i++) {
    matrixMul<<<blocks, threads>>>(d_a, d_b, d_c);
  }

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // Record start event
  hipEventRecord(start);

  for (int i = 0; i < test_round; i++) {
    matrixMul<<<blocks, threads>>>(d_a, d_b, d_c);
  }

  // Record stop event
  hipEventRecord(stop);

  hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  double FLOPs = 2.0 * N * N * N * test_round;
  float GFLOPS = FLOPs / (milliseconds * 1e6);

  printf("Kernel execution time: %.02f ms\n", milliseconds);
  printf("GFLOPS: %.02f gops\n", GFLOPS);

  verify_with_cublas(N, N, N, d_a, d_b, d_c);

  printf("COMPLETED SUCCESSFULLY\n");

  // Free memory on device
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  return 0;
}